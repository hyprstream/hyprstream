
#include <hip/hip_runtime.h>
typedef unsigned int uint;

extern "C" __global__ void kernel(float input_0[], float output_0[],
                                  uint info[]) {

  int threadIdxGlobal = threadIdx.x + threadIdx.y * blockDim.x +
                        threadIdx.z * (blockDim.x * blockDim.y);
  uint rank = info[0];
  uint rank_2 = rank * 2;
  bool l_0_0;
  float l_0_1;
  l_0_0 = threadIdxGlobal == uint(0);
  if (l_0_0) {
    const uint slice_1_0_length = uint(3) - uint(2);
    float *slice_1_0 = output_0 + uint(2);
    uint l_1_0;
    l_1_0 = info[(2 * 2 * info[0]) + 1];
    l_0_1 = (uint(0) < l_1_0) ? input_0[uint(0)] : float(0);
    uint l_1_1;
    bool l_1_2;
    l_1_1 = slice_1_0_length;
    l_1_2 = uint(0) < l_1_1;
    if (l_1_2) {
      slice_1_0[uint(0)] = l_0_1;
    }
  }
}